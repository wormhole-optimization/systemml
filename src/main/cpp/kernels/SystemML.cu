/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/**********************************
When updating a kernel or adding a new one,
please compile the ptx file and commit it:
nvcc -ptx -arch=sm_30 SystemML.cu
***********************************/


#include <hip/hip_runtime.h>
#include <cfloat>
#include <cmath>

/**
 * Performs a slice operation where the input matrix is sparse and the output matrix is dense.
 * This function avoids unnecessary sparse to dense conversion of the input matrix.
 * Parallelization: rows of output matrix.
 * 
 * @params inVal input val pointer
 * @params inRowPtr input row pointer
 * @params colInd input col index pointer
 * @params ret dense output pointer
 * @param rl row lower
 * @param ru row upper
 * @param cl column lower
 * @param cu column upper
 * @param retClen number of columns of output matrix
 */
extern "C"
__global__ void slice_sparse_dense_row(double* inVal, int* inRowPtr, int* colInd, double* ret, 
    int rl, int ru, int cl, int cu, int retClen) {
  	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int rowIndex = index + rl;
  	if (rowIndex <= ru){
  		/*
		 * TODO: Alternative approach: use dynamic parallelism. We are skipping this for now to avoid
		 * the complexity of two-step separate compilation and linking process.
		 *  
		 * extern "C"
		 * __global__ void slice_sparse_dense_row_helper(double* inVal, int* inRowPtr, int* colInd, double* ret, 
		 *     int rl, int ru, int cl, int cu, int retClen, int start, int end, int index) {
		 *  int i = blockIdx.x * blockDim.x + threadIdx.x + start;   
		 * 	// Only slice if the index falls into the given range
		 * 	if(i < end && cl <= colInd[i] && colInd[i] <= cu) {
		 * 		ret[ index*retClen + (colInd[i] - cl) ] = inVal[i];
		 * 	}
		 * }
		 *
		 * int size = inRowPtr[rowIndex+1] - inRowPtr[rowIndex];
		 * double numThreads = (double)min(size, MAX_NUM_THREADS_CHILD_KERNEL);
		 * slice_sparse_dense_row_helper<<< ceil(numThreads/ MAX_NUM_THREADS_CHILD_KERNEL), MAX_NUM_THREADS_CHILD_KERNEL>>>(inVal, inRowPtr, colInd, ret, 
    	 *			rl, ru, cl, cu, retClen, inRowPtr[rowIndex], inRowPtr[rowIndex+1], index);
    	 *
    	 * Two-step compilation and linking process in JCudaKernels's constructor:
    	 * cuLinkAddFile(linkState, CUjitInputType.CU_JIT_INPUT_LIBRARY, "/usr/local/cuda/lib64/libcudadevrt.a", jitOptions);
		 */
    	// Iterate over elements of the row 'rowIndex'.
    	for(int i = inRowPtr[rowIndex]; i < inRowPtr[rowIndex+1]; i++) {
    		// Only slice if the index falls into the given range
    		if(cl <= colInd[i] && colInd[i] <= cu) {
    			ret[ index*retClen + (colInd[i] - cl) ] = inVal[i];
    		}
    	}
    }
}

/**
 * Performs a slice operation where the input matrix is sparse and the output matrix is dense.
 * This function avoids unnecessary sparse to dense conversion of the input matrix.
 * Parallelization: subset of number of non-zeroes of input matrix.
 * 
 * @params inVal input val pointer
 * @params inRowPtr input row pointer
 * @params colInd input col index pointer
 * @params ret dense output pointer
 * @param rl row lower
 * @param ru row upper
 * @param cl column lower
 * @param cu column upper
 * @param retClen number of columns of output matrix
 */
extern "C"
__global__ void slice_sparse_dense_nnz(double* inVal, int* inRowPtr, int* colInd, double* ret, 
    int rl, int ru, int cl, int cu, int retClen) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int i = tid + inRowPtr[rl];
    
    // Only slice if the index falls into the given range
    if(i < inRowPtr[ru+1] && cl <= colInd[i] && colInd[i] <= cu) {
    	// Find the row index for corresponding non-zero value 'i'.
    	int rowIndex = rl;
    	while(inRowPtr[rowIndex+1] <= i) {
    		rowIndex++;
    	}
	    ret[ (rowIndex-rl)*retClen + (colInd[i] - cl) ] = inVal[i];
    }
}

/**
 * Performs a slice operation where the input matrix is dense and the output matrix is dense.
 * 
 * @params in dense input pointer
 * @params ret dense output pointer
 * @param rl row lower
 * @param ru row upper
 * @param cl column lower
 * @param cu column upper
 * @param inClen number of columns of input matrix
 * @param retRlen number of rows of output matrix
 * @param retClen number of columns of output matrix
 */
extern "C"
__global__ void slice_dense_dense(double* in, double* ret, int rl, int ru, int cl, int cu, int inClen, int retRlen, int retClen) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / retClen;
	int iy = tid % retClen;
	if(ix < retRlen && iy < retClen) {
	    int inIndex = (ix + rl)*inClen + cl + iy;
		ret[tid] = in[inIndex];
	}
}


/**
 * Does a copy of upper to lower triangle of the given matrix
 * @param ret the input and output array allocated on the GPU
 * @param dim the number of rows of the square matrix ret
 * @param N total number of elements of the matrix
 */
extern "C"
__global__ void copy_u2l_dense(double* ret, int dim, int N) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / dim;
	int iy = tid % dim;
	int id_dest = iy * dim + ix;
	if(iy > ix && id_dest < N) {
		// TODO: Potential to reduce the number of threads by half
		int id_src = tid;
		ret[id_dest] = ret[id_src];
	}
}

extern "C"
__forceinline__ __device__ double getBoolean(int val) {
	if(val == 0)
		return 0.0;
	else
		return 1.0;
}

// op = {0=plus, 1=minus, 2=multiply, 3=divide, 4=power,
// 5=less, 6=lessequal, 7=greater, 8=greaterequal, 9=equal, 10=notequal,
// 11=min, 12=max, 13=and, 14=or, 15=minus1multiply, 16=minusnz,
// 17=modulus, 18=integer division}
extern "C"
__forceinline__ __device__ double binaryOp(double x, double y, int op) {
	switch(op) {
        case 0 : return x + y;
        case 1 : return x - y;
        case 2 : return x * y;
        case 3 : return x / y;
        case 4 : return pow(x, y);
        case 5 : return getBoolean(x < y);
        case 6 : return getBoolean(x <= y);
        case 7 : return getBoolean(x > y);
        case 8 : return getBoolean(x >= y);
        case 9 : return getBoolean(x == y);
        case 10 : return getBoolean(x != y);
        case 11 : return min(x, y);
        case 12 : return max(x, y);
        case 13 : return getBoolean((int)llrint(x) & (int)llrint(y));
        case 14 : return getBoolean((int)llrint(x) | (int)llrint(y));
        case 15 : return 1 - x * y;
        case 16 : return (x != 0.0 ? x - y : 0.0);
        case 17 : {
            if (y == 0.0 || y == -0.0){
                return nan("");
            }
            double v = x / y;
            // Check for v being NaN (v != v) or if it is infinity
            if (isnan(v) || isinf(v)){
                return v;
            } else {
                v = floor(v);
            }
            return x - v * y;
        }
        case 18:{
            double v = x / y;
            if (isnan(v) || isinf(v)){
                return v;
            } else {
                return floor(v);
            }
        }
        default : return DBL_MAX;
    }
}

extern "C"
__global__ void relu(double* A,  double* ret, int rlen, int clen) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / clen;
	int iy = tid % clen;
	if(ix < rlen && iy < clen) {
		ret[tid] = max(0.0, A[tid]);
	}
}

// This method computes the backpropagation errors for previous layer of relu operation
extern "C"
__global__ void relu_backward(double* X,  double* dout, double* ret, int rlen, int clen) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / clen;
	int iy = tid % clen;
	if(ix < rlen && iy < clen) {
		ret[tid] = X[tid] > 0 ?  dout[tid] : 0;
	}
}

/**
 * Performs inplace addition: ret += input
 *
 * @param input rhs input array allocated on the GPU
 * @param ret the input and output array allocated on the GPU
 * @param rlen the number of rows
 * @param clen the number of columns
 */
extern "C"
__global__ void inplace_add(double* input,  double* ret, int rlen, int clen) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / clen;
	int iy = tid % clen;
	if(ix < rlen && iy < clen) {
		ret[tid] += input[tid];
	}
}

// Performs the operation corresponding to the DML script:
// ones = matrix(1, rows=1, cols=Hout*Wout)
// output = input + matrix(bias %*% ones, rows=1, cols=F*Hout*Wout)
// This operation is often followed by conv2d and hence we have introduced bias_add(input, bias) built-in function
extern "C"
__global__ void bias_add(double* input,  double* bias, double* ret, int rlen, int clen, int PQ) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / clen;
	int iy = tid % clen;
	if(ix < rlen && iy < clen) {
		int biasIndex = iy / PQ;
		ret[tid] = input[tid] + bias[biasIndex];
	}
}

// Performs the operation "ret <- A + alpha*B", where B is a vector
extern "C"
__global__ void daxpy_matrix_vector(double* A,  double* B, double alpha, double* ret, int rlenA, int clenA, int rlenB, int clenB) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / clenA;
	int iy = tid % clenA;
	if(ix < rlenA && iy < clenA) {
		int index = ix * clenA + iy;
		if(rlenB == 1) {
			ret[index] = A[index] + alpha*B[iy];
		}
		else {
			ret[index] = A[index] + alpha*B[ix];
		}
	}
}

// Performs similar operation as bias_add except elementwise multiplication instead of add
extern "C"
__global__ void bias_multiply(double* input,  double* bias, double* ret, int rlen, int clen, int PQ) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / clen;
	int iy = tid % clen;
	if(ix < rlen && iy < clen) {
		int biasIndex = iy / PQ;
		ret[tid] = input[tid] * bias[biasIndex];
	}
}

// Compares the value and set
extern "C"
__global__ void compare_and_set(double* A,  double* ret, int rlen, int clen, double compareVal, double tol, double ifEqualsVal, double ifLessThanVal, double ifGreaterThanVal) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / clen;
	int iy = tid % clen;
	int index = ix * clen + iy;
	if(ix < rlen && iy < clen) {
		if(abs(A[index]-compareVal) < tol)
			ret[index] = ifEqualsVal;
		else if(A[index] < compareVal)
			ret[index] = ifLessThanVal;
		else
			ret[index] = ifGreaterThanVal;
	}
}


/**
 * Performs a binary cellwise arithmetic operation on 2 matrices.
 * Either both matrices are of equal size or one of them is a vector or both are.
 * @param A                 first input matrix allocated on GPU
 * @param B                 second input matrix allocated on GPU
 * @param C                 output allocated on GPU
 * @param maxRlen           maximum of the row lengths of A and B
 * @param maxClen           maximum of the column lengths of A and B
 * @param vectorAStatus     if A is a row vector, column vector or neither
 * @param vectorBStatus     if B is a row vector, column vector or neither
 * @param op                the numeric code of the arithmetic operation to perform
 *
 */
extern "C"
__global__ void matrix_matrix_cellwise_op(double* A, double* B, double* C,
	int maxRlen, int maxClen, int vectorAStatus, int vectorBStatus, int op) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / maxClen;
	int iy = tid % maxClen;

	if(ix < maxRlen && iy < maxClen) {
		int outIndex = ix * maxClen + iy;
		int aIndex = outIndex;
		int bIndex = outIndex;
		if(vectorAStatus == 1)
			aIndex = ix; // clen == 1
		else if(vectorAStatus == 2)
			aIndex = iy; // rlen == 1
		if(vectorBStatus == 1)
			bIndex = ix; // clen == 1
		else if(vectorBStatus == 2)
			bIndex = iy; // rlen == 1
		C[outIndex] = binaryOp(A[aIndex], B[bIndex], op);
		//printf("C[%d] = A[%d](%f) B[%d](%f) (%d %d)\n", outIndex, aIndex, A[aIndex], bIndex,  B[bIndex], (ix+1), (iy+1));
	__syncthreads();
	}
}

/**
 * Performs an arithmetic operation between a matrix and a scalar.
 * C = s op A or C = A op s (where A is the matrix, s is the scalar and op is the operation)
 * @param A             input matrix allocated on GPU
 * @param scalar        scalar input
 * @param C             output matrix allocated on GPU
 * @param size          number of elements in matrix A
 * @param op            number code of the arithmetic operation to perform
 * @param isLeftScalar  whether the scalar is on the left side
 */
extern "C"
__global__ void matrix_scalar_op(double* A, double scalar, double* C, int size, int op, int isLeftScalar) {
	int index = blockIdx.x *blockDim.x + threadIdx.x;
	if(index < size) {
		if(isLeftScalar) {
			C[index] = binaryOp(scalar, A[index], op);
		} else {
			C[index] = binaryOp(A[index], scalar, op);
		}
	}
	__syncthreads();
}


/**
 * Sets all elements (fills) of a double array of given length with a given scalar value
 * @param A         array to be filled
 * @param scalar    value to fill array with
 * @param lenA      length of array A
 */
extern "C"
__global__ void fill(double* A, double scalar, int lenA) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < lenA){
	    A[index] = scalar;
	}
}

/**
 * Appends Matrix B to the right side of Matrix A into a new matrix C
 *         | 1 2 3 4 |   | 8 8 8 |     | 1 2 3 4 8 8 8 |
 * cbind ( | 9 8 7 6 | , | 7 7 7 | ) = | 9 8 7 6 7 7 7 |
 *         | 4 3 2 1 |   | 9 9 9 |     | 4 3 2 1 9 9 9 |
 * @param A      input matrix A allocated on the GPU
 * @param B      input matrix B allocated on the GPU
 * @param C      input matrix C allocated on the GPU
 * @param rowsA  rows in A
 * @param colsA  columns in A
 * @param rowsB  rows in B
 * @param colsB  columns in B
 */
extern "C"
__global__ void cbind(double *A, double *B, double *C, int rowsA, int colsA, int rowsB, int colsB) {
	int maxClen = max(colsA, colsB);
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / maxClen;
	int iy = tid % maxClen;

	int colsC = colsA + colsB;
	int rowsC = rowsA;

	// Copy an element of A into C into the appropriate location
	if (ix < rowsA && iy < colsA) {
		double elemA = A[ix * colsA + iy];
		C[ix * colsC + iy] = elemA;
	}

	// Copy an element of B into C into the appropriate location
	if (ix < rowsB && iy < colsB) {
		double elemB = B[ix * colsB + iy];
		C[ix * colsC + (iy + colsA)] = elemB;
	}
}


/**
 * Appends Matrix B to the bottom of Matrix A into a new matrix C
 *         | 2 3 4 |   | 8 8 8 |     | 2 3 4 |
 * rbind ( | 8 7 6 | , | 7 7 7 | ) = | 8 7 6 |
 *         | 3 2 1 |                 | 3 2 1 |
                                     | 8 8 8 |
                                     | 7 7 7 |
 * @param A      input matrix A allocated on the GPU
 * @param B      input matrix B allocated on the GPU
 * @param C      input matrix C allocated on the GPU
 * @param rowsA  rows in A
 * @param colsA  columns in A
 * @param rowsB  rows in B
 * @param colsB  columns in B
 */
extern "C"
__global__ void rbind(double *A, double *B, double *C, int rowsA, int colsA, int rowsB, int colsB) {
	int maxClen = max(colsA, colsB);
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / maxClen;
	int iy = tid % maxClen;

	int rowsC = rowsA + rowsB;
	int colsC = colsA;

	// Copy an element of A into C into the appropriate location
	if (ix < rowsA && iy < colsA) {
		double elemA = A[ix * colsA + iy];
		C[ix * colsC + iy] = elemA;
	}

	// Copy an element of B into C into the appropriate location
	if (ix < rowsB && iy < colsB) {
		double elemB = B[ix * colsB + iy];
		C[(ix + rowsA) * colsC + iy] = elemB;
	}
}


/**
 * Does a reduce operation over all elements of the array.
 * This method has been adapted from the Reduction sample in the NVIDIA CUDA Samples (v8.0)
 * and the Reduction example available through jcuda.org
 * When invoked initially, all blocks partly compute the reduction operation over the entire array
 * and writes it to the output/temporary array. A second invokation needs to happen to get the
 * reduced value.
 * The number of threads, blocks and amount of shared memory is calculated in a specific way.
 * Please refer to the NVIDIA CUDA Sample or the SystemML code that invokes this method to see
 * how its done.
 * The template-ized version of this function is similar to what is found in NVIDIA CUB
 *
 * @param ReductionOp       Type of the functor object that implements the reduction operation
 */
template <typename ReductionOp>
__device__ void reduce(
    double *g_idata,            ///< input data stored in device memory (of size n)
    double *g_odata,            ///< output/temporary array stored in device memory (of size n)
    unsigned int n,             ///< size of the input and temporary/output arrays
    ReductionOp reduction_op,	///< Reduction operation to perform (functor object)
	double initialValue)  		///< initial value for the reduction variable
{
    extern __shared__ double sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x*2 + threadIdx.x;
    unsigned int gridSize = blockDim.x*2*gridDim.x;

    double v = initialValue;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        v = reduction_op(v, g_idata[i]);
        // ensure we don't read out of bounds
        if (i + blockDim.x < n)
            v = reduction_op(v, g_idata[i+blockDim.x]);
        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = v;
    __syncthreads();


    // do reduction in shared mem
		if (blockDim.x >= 1024){ if (tid < 512) { sdata[tid] = v = reduction_op(v, sdata[tid + 512]); } __syncthreads(); }
    if (blockDim.x >= 512) { if (tid < 256) { sdata[tid] = v = reduction_op(v, sdata[tid + 256]); } __syncthreads(); }
    if (blockDim.x >= 256) { if (tid < 128) { sdata[tid] = v = reduction_op(v, sdata[tid + 128]); } __syncthreads(); }
    if (blockDim.x >= 128) { if (tid <  64) { sdata[tid] = v = reduction_op(v, sdata[tid +  64]); } __syncthreads(); }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile double* smem = sdata;
        if (blockDim.x >=  64) { smem[tid] = v = reduction_op(v, smem[tid + 32]); }
        if (blockDim.x >=  32) { smem[tid] = v = reduction_op(v, smem[tid + 16]); }
        if (blockDim.x >=  16) { smem[tid] = v = reduction_op(v, smem[tid +  8]); }
        if (blockDim.x >=   8) { smem[tid] = v = reduction_op(v, smem[tid +  4]); }
        if (blockDim.x >=   4) { smem[tid] = v = reduction_op(v, smem[tid +  2]); }
        if (blockDim.x >=   2) { smem[tid] = v = reduction_op(v, smem[tid +  1]); }
    }

    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}



/**
 * Does a reduce (sum) over each row of the array.
 * This kernel must be launched with as many blocks as there are rows.
 * The intuition for this kernel is that each block does a reduction over a single row.
 * The maximum number of blocks that can launched (as of compute capability 3.0) is 2^31 - 1
 * This works out fine for SystemML, since the maximum elements in a Java array can be 2^31 - c (some small constant)
 * If the matrix is "fat" and "short", i.e. there are small number of rows and a large number of columns,
 * there could be under-utilization of the hardware.
 * The template-ized version of this function is similar to what is found in NVIDIA CUB
 * @param ReductionOp       Type of the functor object that implements the reduction operation
 * @param AssignmentOp      Type of the functor object that is used to modify the value before writing it to its final location in global memory for each row
 */
template <typename ReductionOp,
          typename AssignmentOp>
__device__ void reduce_row(
    double *g_idata,            ///< input data stored in device memory (of size rows*cols)
    double *g_odata,            ///< output/temporary array store in device memory (of size rows*cols)
    unsigned int rows,          ///< rows in input and temporary/output arrays
    unsigned int cols,          ///< columns in input and temporary/output arrays
    ReductionOp reduction_op,		///< Reduction operation to perform (functor object)
    AssignmentOp assignment_op, ///< Operation to perform before assigning this to its final location in global memory for each row
    double initialValue){  			///< initial value for the reduction variable
    extern __shared__ double sdata[];

    // one block per row
    if (blockIdx.x >= rows) {
        return;
    }

    unsigned int block = blockIdx.x;
    unsigned int tid = threadIdx.x;
    unsigned int i = tid;
    unsigned int block_offset = block * cols;

    double v = initialValue;
    while (i < cols){
        v = reduction_op(v, g_idata[block_offset + i]);
        i += blockDim.x;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = v;
    __syncthreads();

 		// do reduction in shared mem
  	if (blockDim.x >= 1024){ if (tid < 512) { sdata[tid] = v = reduction_op(v, sdata[tid + 512]); } __syncthreads(); }
    if (blockDim.x >= 512) { if (tid < 256) { sdata[tid] = v = reduction_op(v, sdata[tid + 256]); } __syncthreads(); }
    if (blockDim.x >= 256) { if (tid < 128) { sdata[tid] = v = reduction_op(v, sdata[tid + 128]); } __syncthreads(); }
    if (blockDim.x >= 128) { if (tid <  64) { sdata[tid] = v = reduction_op(v, sdata[tid +  64]); } __syncthreads(); }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile double* smem = sdata;
        if (blockDim.x >=  64) { smem[tid] = v = reduction_op(v, smem[tid + 32]); }
        if (blockDim.x >=  32) { smem[tid] = v = reduction_op(v, smem[tid + 16]); }
        if (blockDim.x >=  16) { smem[tid] = v = reduction_op(v, smem[tid +  8]); }
        if (blockDim.x >=   8) { smem[tid] = v = reduction_op(v, smem[tid +  4]); }
        if (blockDim.x >=   4) { smem[tid] = v = reduction_op(v, smem[tid +  2]); }
        if (blockDim.x >=   2) { smem[tid] = v = reduction_op(v, smem[tid +  1]); }
    }

    // write result for this block to global mem, modify it with assignment op
    if (tid == 0)
        g_odata[block] = assignment_op(sdata[0]);
}


/**
 * Does a column wise reduction.
 * The intuition is that there are as many global threads as there are columns
 * Each global thread is responsible for a single element in the output vector
 * This of course leads to a under-utilization of the GPU resources.
 * For cases, where the number of columns is small, there can be unused SMs
 *
 * The template-ized version of this function is similar to what is found in NVIDIA CUB
 * @param ReductionOp       Type of the functor object that implements the reduction operation
 * @param AssignmentOp      Type of the functor object that is used to modify the value before writing it to its final location in global memory for each column
 */
template <typename ReductionOp,
          typename AssignmentOp>
__device__ void reduce_col(
    double *g_idata,            ///< input data stored in device memory (of size rows*cols)
    double *g_odata,            ///< output/temporary array store in device memory (of size rows*cols)
    unsigned int rows,          ///< rows in input and temporary/output arrays
    unsigned int cols,          ///< columns in input and temporary/output arrays
    ReductionOp reduction_op,	///< Reduction operation to perform (functor object)
    AssignmentOp assignment_op, ///< Operation to perform before assigning this to its final location in global memory for each column
    double initialValue)  		///< initial value for the reduction variable
{
    unsigned int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_tid >= cols) {
        return;
    }

    unsigned int i = global_tid;
    unsigned int grid_size = cols;
    double val = initialValue;

    while (i < rows * cols) {
      val = reduction_op(val, g_idata[i]);
      i += grid_size;
    }
    g_odata[global_tid] = assignment_op(val);
}

/**
 * Functor op for assignment op. This is a dummy/identity op.
 */
typedef struct {
    __device__ __forceinline__
    double operator()(double a) const {
        return a;
    }
} IdentityOp;

/**
 * Functor op for summation operation
 */
typedef struct {
    __device__ __forceinline__
    double operator()(double a, double b) const {
        return a + b;
    }
} SumOp;


/**
 * Do a summation over all elements of an array/matrix
 * @param g_idata   input data stored in device memory (of size n)
 * @param g_odata   output/temporary array stored in device memory (of size n)
 * @param n         size of the input and temporary/output arrays
 */
extern "C"
__global__ void reduce_sum(double *g_idata, double *g_odata, unsigned int n){
	SumOp op;
  reduce<SumOp>(g_idata, g_odata, n, op, 0.0);
}

/**
 * Do a summation over all rows of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size rows)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
extern "C"
__global__ void reduce_row_sum(double *g_idata, double *g_odata, unsigned int rows, unsigned int cols){
    SumOp op;
    IdentityOp aop;
    reduce_row<SumOp, IdentityOp>(g_idata, g_odata, rows, cols, op, aop, 0.0);
}

/**
 * Do a summation over all columns of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size cols)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
extern "C"
__global__ void reduce_col_sum(double *g_idata, double *g_odata, unsigned int rows, unsigned int cols){
    SumOp op;
    IdentityOp aop;
    reduce_col<SumOp, IdentityOp>(g_idata, g_odata, rows, cols, op, aop, 0.0);
}


/**
 * Functor op for max operation
 */
typedef struct {
    __device__ __forceinline__
    double operator()(double a, double b) const {
        return fmax(a, b);
    }
} MaxOp;


/**
 * Do a max over all elements of an array/matrix
 * @param g_idata   input data stored in device memory (of size n)
 * @param g_odata   output/temporary array stode in device memory (of size n)
 * @param n         size of the input and temporary/output arrays
 */
extern "C"
__global__ void reduce_max(double *g_idata, double *g_odata, unsigned int n){
    MaxOp op;
    reduce<MaxOp>(g_idata, g_odata, n, op, -DBL_MAX);
}

/**
 * Do a max over all rows of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size rows)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
extern "C"
__global__ void reduce_row_max(double *g_idata, double *g_odata, unsigned int rows, unsigned int cols){
    MaxOp op;
    IdentityOp aop;
    reduce_row<MaxOp, IdentityOp>(g_idata, g_odata, rows, cols, op, aop, -DBL_MAX);
}

/**
 * Do a max over all columns of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size cols)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
extern "C"
__global__ void reduce_col_max(double *g_idata, double *g_odata, unsigned int rows, unsigned int cols){
    MaxOp op;
    IdentityOp aop;
    reduce_col<MaxOp, IdentityOp>(g_idata, g_odata, rows, cols, op, aop, -DBL_MAX);
}

/**
 * Functor op for min operation
 */
typedef struct {
    __device__ __forceinline__
    double operator()(double a, double b) const {
        return fmin(a, b);
    }
} MinOp;

/**
 * Do a min over all elements of an array/matrix
 * @param g_idata   input data stored in device memory (of size n)
 * @param g_odata   output/temporary array stode in device memory (of size n)
 * @param n         size of the input and temporary/output arrays
 */
extern "C"
__global__ void reduce_min(double *g_idata, double *g_odata, unsigned int n){
	MinOp op;
    reduce<MinOp>(g_idata, g_odata, n, op, DBL_MAX);
}

/**
 * Do a min over all rows of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size rows)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
extern "C"
__global__ void reduce_row_min(double *g_idata, double *g_odata, unsigned int rows, unsigned int cols){
    MinOp op;
    IdentityOp aop;
    reduce_row<MinOp, IdentityOp>(g_idata, g_odata, rows, cols, op, aop, DBL_MAX);
}

/**
 * Do a min over all columns of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size cols)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
extern "C"
__global__ void reduce_col_min(double *g_idata, double *g_odata, unsigned int rows, unsigned int cols){
    MinOp op;
    IdentityOp aop;
    reduce_col<MinOp>(g_idata, g_odata, rows, cols, op, aop, DBL_MAX);
}

/**
 * Functor op for product operation
 */
typedef struct {
    __device__ __forceinline__
    double operator()(double a, double b) const {
        return a * b;
    }
} ProductOp;

/**
 * Do a product over all elements of an array/matrix
 * @param g_idata   input data stored in device memory (of size n)
 * @param g_odata   output/temporary array stode in device memory (of size n)
 * @param n         size of the input and temporary/output arrays
 */
extern "C"
__global__ void reduce_prod(double *g_idata, double *g_odata, unsigned int n){
	ProductOp op;
    reduce<ProductOp>(g_idata, g_odata, n, op, 1.0);
}

/**
 * Functor op for mean operation
 */
struct MeanOp {
    const long _size;   ///< Number of elements by which to divide to calculate mean
		__device__ __forceinline__
    MeanOp(long size): _size(size) {}
    __device__ __forceinline__
    double operator()(double total) const {
        return total / _size;
    }
};


/**
 * Do a mean over all rows of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size rows)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
extern "C"
__global__ void reduce_row_mean(double *g_idata, double *g_odata, unsigned int rows, unsigned int cols){
    SumOp op;
    MeanOp aop(cols);
    reduce_row<SumOp, MeanOp>(g_idata, g_odata, rows, cols, op, aop, 0.0);
}

/**
 * Do a mean over all columns of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size cols)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
extern "C"
__global__ void reduce_col_mean(double *g_idata, double *g_odata, unsigned int rows, unsigned int cols){
    SumOp op;
    MeanOp aop(rows);
    reduce_col<SumOp, MeanOp>(g_idata, g_odata, rows, cols, op, aop, 0.0);
}


/**
 * Do an exp over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
extern "C"
__global__ void matrix_exp(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        C[index] = exp(A[index]);
    }
}

/**
 * Do an sqrt over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
extern "C"
__global__ void matrix_sqrt(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        C[index] = sqrt(A[index]);
    }
}

/**
 * Do an round over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
extern "C"
__global__ void matrix_round(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        C[index] = (double)llround(A[index]);
    }
}

/**
 * Do an abs over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
extern "C"
__global__ void matrix_abs(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        C[index] = (double)fabs(A[index]);
    }
}

/**
 * Do an log over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
extern "C"
__global__ void matrix_log(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        C[index] = log(A[index]);
    }
}

/**
 * Do an floor over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
extern "C"
__global__ void matrix_floor(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        C[index] = floor(A[index]);
    }
}

/**
 * Do an ceil over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
extern "C"
__global__ void matrix_ceil(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        C[index] = ceil(A[index]);
    }
}

/**
 * Do an sin over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
extern "C"
__global__ void matrix_sin(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        C[index] = sin(A[index]);
    }
}

/**
 * Do an sinh over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
extern "C"
__global__ void matrix_sinh(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        C[index] = sinh(A[index]);
    }
}

/**
 * Do an cos over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
extern "C"
__global__ void matrix_cos(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        C[index] = cos(A[index]);
    }
}

/**
 * Do an cosh over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
extern "C"
__global__ void matrix_cosh(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        C[index] = cosh(A[index]);
    }
}

/**
 * Do an tan over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
extern "C"
__global__ void matrix_tan(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        C[index] = tan(A[index]);
    }
}

/**
 * Do an tanh over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
extern "C"
__global__ void matrix_tanh(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        C[index] = tanh(A[index]);
    }
}

/**
 * Do an asin over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
extern "C"
__global__ void matrix_asin(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        C[index] = asin(A[index]);
    }
}

/**
 * Do an acos over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
extern "C"
__global__ void matrix_acos(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        C[index] = acos(A[index]);
    }
}

/**
 * Do an atan over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
extern "C"
__global__ void matrix_atan(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        C[index] = atan(A[index]);
    }
}

/**
 * Do an sign over all the elements of a matrix
 * Assign -1, 0 or 1 depending on the element being negative, 0 or positive
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
extern "C"
__global__ void matrix_sign(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        if (A[index] == 0.0) {
            C[index] = 0.0;
        } else {
            C[index] = copysign(1.0, A[index]);
        }
    }
}
